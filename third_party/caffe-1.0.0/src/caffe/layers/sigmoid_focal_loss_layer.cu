#include "hip/hip_runtime.h"
#include <vector>
#include <cfloat>
#include <algorithm>

#include "caffe/layers/sigmoid_focal_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {


template <typename Dtype>
__global__ void SigmoidFocalLossForwardGPU(const int nthreads, const Dtype* sigmoid_output_data,
          const Dtype* input_data, const Dtype* target, Dtype* loss,
          const bool has_ignore_label_, const int ignore_label_, const bool weight_by_label_freqs_,
          Dtype* counts, const float* label_count_data, const Dtype gamma_) {
  CUDA_KERNEL_LOOP(i, nthreads) {
    const int target_value = static_cast<int>(target[i]);
    if (has_ignore_label_ && target_value == ignore_label_) {
      loss[i] = 0;
      counts[i] = 0;
    } else {
      if(weight_by_label_freqs_){
        if(target_value==0){
          loss[i] = log(1 + exp(input_data[i])) * pow(sigmoid_output_data[i], gamma_)
                  * static_cast<Dtype>(label_count_data[target_value]);
        } else {
          loss[i] = (log(1 + exp(input_data[i])) - input_data[i]) * pow(1 - sigmoid_output_data[i], gamma_)
                  * static_cast<Dtype>(label_count_data[target_value]);
        }
      } else {
        if(target_value==0){
          loss[i] = log(1 + exp(input_data[i])) * pow(sigmoid_output_data[i], gamma_);
        } else {
          loss[i] = (log(1 + exp(input_data[i])) - input_data[i]) * pow(1 - sigmoid_output_data[i], gamma_);
        }
      }
      counts[i] = 1;
    }
  }
}

template <typename Dtype>
void SigmoidFocalLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  // The forward pass computes the sigmoid outputs.
  sigmoid_bottom_vec_[0] = bottom[0];
  sigmoid_layer_->Forward(sigmoid_bottom_vec_, sigmoid_top_vec_);
  const Dtype* sigmoid_output_data = sigmoid_output_->gpu_data();
  // get the class weight
  float* label_count_data=NULL;
  if(weight_by_label_freqs_){
    label_count_data = label_counts_.mutable_gpu_data();
  }
  // Compute the loss (negative log likelihood)
  const int count = bottom[0]->count();
  // Stable version of loss computation from input data
  const Dtype* input_data = bottom[0]->gpu_data();
  const Dtype* target = bottom[1]->gpu_data();
  // Since this memory is not used for anything until it is overwritten
  // on the backward pass, we use it here to avoid having to allocate new GPU
  // memory to accumulate intermediate results in the kernel.
  Dtype* loss_data = bottom[0]->mutable_gpu_diff();
  Dtype* count_data = bottom[1]->mutable_gpu_diff();
  Dtype valid_count=0;
  // NOLINT_NEXT_LINE(whitespace/operators)
  SigmoidFocalLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS>>>(count, sigmoid_output_data, input_data, target, loss_data,
      has_ignore_label_, ignore_label_, weight_by_label_freqs_, count_data, label_count_data, gamma_);
  // Only launch another CUDA kernel if we actually need the valid count.
  if (normalization_ == LossParameter_NormalizationMode_VALID &&
      has_ignore_label_) {
    caffe_gpu_asum(count, count_data, &valid_count);
    // std::cout<< "valid count " << valid_count<<std::endl;
  } else {
    valid_count = count;
  }
  Dtype loss;
  caffe_gpu_asum(count, loss_data, &loss);
  normalizer_ = get_normalizer(normalization_, valid_count);
  // std::cout<< "normalizer " << normalizer_<<std::endl;
  top[0]->mutable_cpu_data()[0] = loss / normalizer_;
}

template <typename Dtype>
__global__ void SigmoidFocalLossBackwardGPU(const int nthreads, const Dtype* sigmoid_output_data,
          const Dtype* target, Dtype* bottom_diff, const bool has_ignore_label_, const int ignore_label_,
          const bool weight_by_label_freqs_, const float* label_count_data, const Dtype gamma_) {
  CUDA_KERNEL_LOOP(i, nthreads) {
    const int target_value = static_cast<int>(target[i]);
    if (has_ignore_label_ && target_value == ignore_label_) {
      // Zero out gradient of ignored targets.
      bottom_diff[i] = 0;
      } else { 
        if (target_value==0) {
          if(weight_by_label_freqs_) {
            bottom_diff[i] = -pow(sigmoid_output_data[i], gamma_)
              *(gamma_*(1-sigmoid_output_data[i])*log(max(1-sigmoid_output_data[i], Dtype(FLT_MIN))) - sigmoid_output_data[i])
              * static_cast<Dtype>(label_count_data[target_value]);
          } else {
            bottom_diff[i] = -pow(sigmoid_output_data[i], gamma_)
              *(gamma_*(1-sigmoid_output_data[i])*log(max(1-sigmoid_output_data[i], Dtype(FLT_MIN))) - sigmoid_output_data[i]);
          }
      } else {
        if(weight_by_label_freqs_) {
          bottom_diff[i] = pow(1-sigmoid_output_data[i], gamma_)
            *(gamma_*sigmoid_output_data[i]*log(max(sigmoid_output_data[i], Dtype(FLT_MIN))) + sigmoid_output_data[i]-1)
            * static_cast<Dtype>(label_count_data[target_value]);
        } else {
          bottom_diff[i] = pow(1-sigmoid_output_data[i], gamma_)
            *(gamma_*sigmoid_output_data[i]*log(max(sigmoid_output_data[i], Dtype(FLT_MIN))) + sigmoid_output_data[i]-1);
        }
      }
    }
  }
}

template <typename Dtype>
void SigmoidFocalLossLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    // get the class weight
    float* label_count_data=NULL;
    if(weight_by_label_freqs_){
      label_count_data = label_counts_.mutable_gpu_data();
    }
    // First, compute the diff
    const int count = bottom[0]->count();
    const Dtype* sigmoid_output_data = sigmoid_output_->gpu_data();
    const Dtype* target = bottom[1]->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    SigmoidFocalLossBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS>>>(count, sigmoid_output_data, target, bottom_diff,
      has_ignore_label_, ignore_label_, weight_by_label_freqs_, label_count_data, gamma_);
    // Scale down gradient
    Dtype loss_weight = top[0]->cpu_diff()[0] / normalizer_;
    caffe_gpu_scal(count, loss_weight, bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SigmoidFocalLossLayer);

}  // namespace caffe
