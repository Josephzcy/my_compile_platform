#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/hardtanh_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void HardtanhForward(const int n, const Dtype* in, Dtype* out,
    Dtype low_bound, Dtype high_bound) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > low_bound ? in[index] : low_bound;
    out[index] = out[index] < high_bound ? out[index] : high_bound;
  }
}

template <typename Dtype>
void HardtanhLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  Dtype low_bound = this->layer_param_.hardtanh_param().low_bound();
  Dtype high_bound = this->layer_param_.hardtanh_param().high_bound();
  // NOLINT_NEXT_LINE(whitespace/operators)
  HardtanhForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data, low_bound, high_bound);
  CUDA_POST_KERNEL_CHECK;
  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data
  //     << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
}

template <typename Dtype>
__global__ void HardtanhBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff, Dtype low_bound, Dtype high_bound) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * (in_data[index] > low_bound) * (in_data[index] < high_bound);
  }
}

template <typename Dtype>
void HardtanhLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
	Dtype low_bound = this->layer_param_.hardtanh_param().low_bound();
	Dtype high_bound = this->layer_param_.hardtanh_param().high_bound();
    // NOLINT_NEXT_LINE(whitespace/operators)
    HardtanhBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, bottom_diff, low_bound, high_bound);
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(HardtanhLayer);


}  // namespace caffe

