#include "hip/hip_runtime.h"
#include "ristretto/base_ristretto_layer.hpp"
#include "ristretto/base_ristretto_layer.cuh"

namespace caffe {

template <typename Dtype>
void BaseRistrettoLayer<Dtype>::QuantizeWeights_gpu(
      vector<shared_ptr<Blob<Dtype> > > weights_quantized, const int rounding,
      const bool bias_term) {
  Dtype* weight = weights_quantized[0]->mutable_gpu_data();
  const int cnt_weight = weights_quantized[0]->count();
  switch (precision_) {
  case QuantizationParameter_Precision_MINIFLOAT:
    Trim2MiniFloat_gpu(weight, cnt_weight, fp_mant_, fp_exp_, rounding);
    if (bias_term) {
      Trim2MiniFloat_gpu(weights_quantized[1]->mutable_gpu_data(),
          weights_quantized[1]->count(), fp_mant_, fp_exp_, rounding);
    }
    break;
  case QuantizationParameter_Precision_DYNAMIC_FIXED_POINT:   //weights always use signed quantization method
    Trim2FixedPoint_gpu(weight, cnt_weight, bw_params_, rounding, fl_params_, false);
    if (bias_term) {
      Trim2FixedPoint_gpu(weights_quantized[1]->mutable_gpu_data(),
          weights_quantized[1]->count(), bw_params_, rounding, fl_params_, false);
    }
    break;
  case QuantizationParameter_Precision_INTEGER_POWER_OF_2_WEIGHTS:
    Trim2IntegerPowerOf2_gpu(weight, cnt_weight, pow_2_min_exp_, pow_2_max_exp_,
        rounding);
    // Don't trim bias
    break;
  default:
    LOG(FATAL) << "Unknown trimming mode: " << precision_;
    break;
  }
}

template <typename Dtype>
void BaseRistrettoLayer<Dtype>::QuantizeLayerInputs_gpu(
    Dtype* data, const int count) {
  switch (precision_) {
    case QuantizationParameter_Precision_INTEGER_POWER_OF_2_WEIGHTS:
    case QuantizationParameter_Precision_DYNAMIC_FIXED_POINT:
      Trim2FixedPoint_gpu(data, count, bw_layer_in_, rounding_, fl_layer_in_, unsigned_in_);
      break;
    case QuantizationParameter_Precision_MINIFLOAT:
      Trim2MiniFloat_gpu(data, count, fp_mant_, fp_exp_, rounding_);
      break;
    default:
      LOG(FATAL) << "Unknown trimming mode: " << precision_;
      break;
  }
}

template <typename Dtype>
void BaseRistrettoLayer<Dtype>::QuantizeLayerOutputs_gpu(Dtype* data,
      const int count) {
  switch (precision_) {
    case QuantizationParameter_Precision_INTEGER_POWER_OF_2_WEIGHTS:
    case QuantizationParameter_Precision_DYNAMIC_FIXED_POINT:
      Trim2FixedPoint_gpu(data, count, bw_layer_out_, rounding_, fl_layer_out_, unsigned_out_);
      break;
    case QuantizationParameter_Precision_MINIFLOAT:
      Trim2MiniFloat_gpu(data, count, fp_mant_, fp_exp_, rounding_);
      break;
    default:
      LOG(FATAL) << "Unknown trimming mode: " << precision_;
      break;
  }
}


template <typename Dtype>
__global__ void Trim2FixedPoint_kernel(Dtype* data, const int cnt,
      const int bit_width, const int rounding, const int fl, const bool unsigned_or_not) {
	CUDA_KERNEL_LOOP(index, cnt) {
    
    // Saturate data
	Dtype max_data, min_data;
    if (unsigned_or_not == false)
	  {
		  max_data = (pow(2.0, bit_width - 1) - 1) * pow(2.0, -fl);
		  min_data = -pow(2.0, bit_width - 1) * pow(2.0, -fl);
	  }
	  else
	  {
		  max_data = (pow(2.0, bit_width) - 1) * pow(2.0, -fl);
		  min_data = 0;
	  }
    data[index] = max(min(data[index], max_data), min_data);
    // Round data
    data[index] /= pow(2.0, -fl);
    switch (rounding) {
    case QuantizationParameter_Rounding_NEAREST:
      	if ((data[index] - __double2int_rd(data[index])) == 0.5 && data[index]<0)
	    {
		     data[index] = round(data[index])+1;
	    }
	    else
	    {
		     data[index] = round(data[index]);
      }
      //data[index] = __double2int_rd(data[index]);
      break;
    case QuantizationParameter_Rounding_STOCHASTIC:
      data[index] = __double2int_rd(data[index] + RandUniform_device(index));
      break;
    default:
      break;
    }
    data[index] *= pow(2.0, -fl);
	}
}

template <typename Dtype>
void BaseRistrettoLayer<Dtype>::Trim2FixedPoint_gpu(Dtype* data, const int cnt,
      const int bit_width, const int rounding, int fl, bool unsigned_or_not) {
  Trim2FixedPoint_kernel<<<CAFFE_GET_BLOCKS(cnt), CAFFE_CUDA_NUM_THREADS>>>(
      data, cnt, bit_width, rounding, fl, unsigned_or_not);
}

template <typename Dtype>
__global__ void Trim2MiniFloat_kernel(Dtype* data, const int cnt,
      const int bw_mant, const int bw_exp, const int rounding){
	CUDA_KERNEL_LOOP(index, cnt) {
    Trim2MiniFloat_device(&data[index], bw_mant, bw_exp, rounding, index);
	}
}

template <typename Dtype>
void BaseRistrettoLayer<Dtype>::Trim2MiniFloat_gpu(Dtype* data,
      const int cnt, const int bw_mant, const int bw_exp, const int rounding) {
  Trim2MiniFloat_kernel<<<CAFFE_GET_BLOCKS(cnt), CAFFE_CUDA_NUM_THREADS>>>(
      data, cnt, bw_mant, bw_exp, rounding);
}

template <typename Dtype>
__global__ void Trim2IntegerPowerOf2_kernel(Dtype* data, const int cnt,
      const int min_exp, const int max_exp, const int rounding) {
	CUDA_KERNEL_LOOP(index, cnt) {
    float exponent = log2f(fabs((float)data[index]));
    int sign = data[index] >= 0 ? 1 : -1;
    switch (rounding) {
    case QuantizationParameter_Rounding_NEAREST:
      exponent = rint(exponent);
      break;
    case QuantizationParameter_Rounding_STOCHASTIC:
      exponent = __float2int_rd(exponent + RandUniform_device(index));
      break;
    default:
      break;
    }
    exponent = fmaxf(fminf(exponent, max_exp), min_exp);
    data[index] = sign * powf(2, exponent);
	}
}

template <typename Dtype>
void BaseRistrettoLayer<Dtype>::Trim2IntegerPowerOf2_gpu(Dtype* data,
      const int cnt, const int min_exp, const int max_exp, const int rounding) {
  Trim2IntegerPowerOf2_kernel<<<CAFFE_GET_BLOCKS(cnt), CAFFE_CUDA_NUM_THREADS>>>(
      data, cnt, min_exp, max_exp, rounding);
}

// Explicit instantiations
template void BaseRistrettoLayer<double>::QuantizeWeights_gpu(
    vector<shared_ptr<Blob<double> > > weights_quantized, const int rounding,
    const bool bias_term);
template void BaseRistrettoLayer<float>::QuantizeWeights_gpu(
    vector<shared_ptr<Blob<float> > > weights_quantized, const int rounding,
    const bool bias_term);
template void BaseRistrettoLayer<double>::QuantizeLayerInputs_gpu(double* data,
    const int count);
template void BaseRistrettoLayer<float>::QuantizeLayerInputs_gpu(float* data,
    const int count);
template void BaseRistrettoLayer<double>::QuantizeLayerOutputs_gpu(
    double* top_data, const int top_count);
template void BaseRistrettoLayer<float>::QuantizeLayerOutputs_gpu(
    float* top_data, const int top_count);
template void BaseRistrettoLayer<double>::Trim2FixedPoint_gpu(double* data,
    const int cnt, const int bit_width, const int rounding, int fl, bool unsigned_flag);
template void BaseRistrettoLayer<float>::Trim2FixedPoint_gpu(float* data,
    const int cnt, const int bit_width, const int rounding, int fl, bool unsigned_flag);
template void BaseRistrettoLayer<double>::Trim2MiniFloat_gpu(double* data,
    const int cnt, const int bw_mant, const int bw_exp, const int rounding);
template void BaseRistrettoLayer<float>::Trim2MiniFloat_gpu(float* data,
    const int cnt, const int bw_mant, const int bw_exp, const int rounding);
template void BaseRistrettoLayer<double>::Trim2IntegerPowerOf2_gpu(double* data,
    const int cnt, const int min_exp, const int max_exp, const int rounding);
template void BaseRistrettoLayer<float>::Trim2IntegerPowerOf2_gpu(float* data,
    const int cnt, const int min_exp, const int max_exp, const int rounding);

}  // namespace caffe


